#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optixPathTracer.h"
#include "random.h"
#include <optix_math.h>

using namespace optix;

rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );
rtDeclareVariable(float3, diffuse_color, , );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(unsigned int, pathtrace_ray_type, , );
rtDeclareVariable(unsigned int, pathtrace_shadow_ray_type, , );
rtDeclareVariable(unsigned int, pathtrace_insect_ray_type, , );
rtDeclareVariable(float3, bg_color, , );

rtBuffer<ParallelogramLight>     lights;



RT_PROGRAM void diffuse()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hitpoint = ray.origin + t_hit * ray.direction;


	current_prd.attenuation = current_prd.attenuation * diffuse_color;
	current_prd.countEmitted = false;


	unsigned int num_lights = lights.size();
	float3 result = make_float3(0.0f);

	for (int i = 0; i < num_lights; ++i)
	{
		// Choose random point on light
		ParallelogramLight light = lights[i];
		const float z1 = rnd(current_prd.seed);
		const float z2 = rnd(current_prd.seed);
		const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;
		//const float3 light_pos = light.corner + light.v1 * 0.5 + light.v2 * 0.5;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = dot(light.normal, L);

		// cast shadow ray
		if (nDl > 0.0f && LnDl > 0.0f)
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = false;
			Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
			rtTrace(top_shadower, shadow_ray, shadow_prd);

			if (!shadow_prd.inShadow)
			{
				const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
				result += light.emission * weight;
			}
		}
		/*PerRayData_pathtrace_insect insect_prd;
		Ray insect_ray = make_Ray(light_pos, -L, pathtrace_insect_ray_type, scene_epsilon, Ldist - scene_epsilon);
		rtTrace(top_shadower, insect_ray, insect_prd);
		double reducenum = exp(1 + length(insect_prd.hitpoint - hitpoint) / 100);
		result += insect_prd.radiance / reducenum;*/
	}

	int scatternum = 15;
	for (int i = 0; i < scatternum; ++i)
	{
		float z1 = rnd(current_prd.seed);
		float z2 = rnd(current_prd.seed);
		float3 p1;
		cosine_sample_hemisphere(z1, z2, p1);
		optix::Onb onb1(-ffnormal);
		onb1.inverse_transform(p1);

		PerRayData_pathtrace_insect insect_prd;
		Ray insect_ray = make_Ray(hitpoint, p1, pathtrace_insect_ray_type, scene_epsilon, 0x7f7f7f7f);
		rtTrace(top_shadower, insect_ray, insect_prd);
		double reducenum = exp(1 + length(insect_prd.hitpoint - hitpoint) / 75);
		result += insect_prd.radiance / reducenum;
	}


	current_prd.radiance = result;


	/*double sigmaT = 10.01f;
	double bias = 0.99;
	double ul = bias * rnd(current_prd.seed);
	double r_max = -(1.0f / sigmaT) * log(1.0f - bias);
	double r = -(1.0f / sigmaT) * log(1.0f - ul);
	double l = sqrtf(r_max*r_max - r*r);

	float ztmp = rnd(current_prd.seed);
	float3 p = make_float3(r*cos(ztmp), r*sin(ztmp), -l);
	optix::Onb onb(ffnormal);
	onb.inverse_transform(p);

	float3 base_pos = hitpoint + p;
	float3 pTarget = base_pos - l * ffnormal;
	
	PerRayData_pathtrace_insect insect_prd;
	Ray insect_ray = make_Ray(base_pos, -ffnormal, pathtrace_insect_ray_type, scene_epsilon, r_max);
	rtTrace(top_shadower, insect_ray, insect_prd);
	
	
	current_prd.origin = insect_prd.hitpoint;

	float z1 = rnd(current_prd.seed);
	float z2 = rnd(current_prd.seed);
	float3 p1;
	cosine_sample_hemisphere(z1, z2, p1);
	optix::Onb onb1(insect_prd.normal);
	onb1.inverse_transform(p1);
	current_prd.direction = p1;

	

	bool into = dot(ffnormal, world_geometric_normal) > 0.0;
	double nc = 1.0;
	double nt = 1.3;
	double nnt = into ? nc / nt : nt / nc;
	double ddn = dot(ray.direction, ffnormal);
	double a = nt - nc, b = nt + nc;
	double R0 = (a * a) / (b * b);
	double c = 1.0 + ddn;
	double Re_in = R0 + (1.0 - R0) * pow(c, 5.0);
	double Tr_in = 1.0 - Re_in;

	nnt = 1 / nnt;
	c = 1.0 - dot(current_prd.direction, ffnormal);
	const double Re_out = R0 + (1.0 - R0) * pow(c, 5.0);
	const double Tr_out = 1.0 - Re_out;

	double albed_dush = 0.999001;
	double sigma_tr = 0.547996;
	double zr = 0.0999;
	double zv = -0.446566;
	double r2 = r * r;
	double dr = sqrtf(r2 + zr * zr);
	double dv = sqrtf(r2 + zv * zv);
	double phi_r = zr * (dr * sigma_tr + 1) * exp(-sigma_tr * dr) / (dr * dr * dr);
	double phi_v = zv * (dv * sigma_tr + 1) * exp(-sigma_tr * dv) / (dv * dv * dv);
	double Rd = (albed_dush / (4.0 * 3.1415926)) * (phi_r - phi_v);
	float Sd = Tr_out*(1.0 / 3.1415926)*Tr_in;
	current_prd.radiance = result + insect_prd.radiance;*/
}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
	current_prd_shadow.inShadow = true;
	rtTerminateRay();
}



// Insect close hit

rtDeclareVariable(PerRayData_pathtrace_insect, current_prd_insect, rtPayload, );

RT_PROGRAM void find_insect()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = world_geometric_normal;

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	current_prd_insect.hitpoint = hitpoint;
	current_prd_insect.normal = ffnormal;

	unsigned int num_lights = lights.size();
	float3 result = make_float3(0.0f);

	for (int i = 0; i < num_lights; ++i)
	{
		// Choose random point on light
		ParallelogramLight light = lights[i];
		const float z1 = rnd(current_prd.seed);
		const float z2 = rnd(current_prd.seed);
		const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = dot(light.normal, L);

		// cast shadow ray
		//if (nDl > 0.0f && LnDl > 0.0f)
		//{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = false;
			Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
			rtTrace(top_shadower, shadow_ray, shadow_prd);

			if (!shadow_prd.inShadow)
			{
				const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
				result += light.emission * fabs(weight);
			}
		//}
	}
	current_prd_insect.radiance = result;
}
